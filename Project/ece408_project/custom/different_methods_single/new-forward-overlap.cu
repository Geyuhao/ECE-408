#include "hip/hip_runtime.h"
#include <cmath>
#include <iostream>
#include "gpu-new-forward.h"

# define TILE_WIDTH 8
# define new_TILE_WIDTH 16

__constant__ float MASK[7*7*16*4];

/*
Modify this function to implement the forward pass described in Chapter 16.
We have added an additional dimension to the tensors to support an entire mini-batch
The goal here is to be correct AND fast.

Function paramter definitions:
y - output
x - input
k - kernel
B - batch_size (number of images in x)      100/1000/10000
M - number of output feature maps           4   16
C - number of input feature maps            1   4
H - input height dimension                  86  40
W - input width dimension                   86  40
K - kernel height and width (K x K)         7   7
*/

__global__ void conv_forward_kernel(float *y, const float *x, const float *k, const int B, const int M, const int C, const int H, const int W, const int K)
{
    const int H_out = H - K + 1;
    const int W_out = W - K + 1;
    //(void)H_out; // silence declared but never referenced warning. remove this line when you start working
    //(void)W_out; // silence declared but never referenced warning. remove this line when you start working

    // We have some nice #defs for you below to simplify indexing. Feel free to use them, or create your own.
    // An example use of these macros:
    // float a = y4d(0,0,0,0)
    // y4d(0,0,0,0) = a

    #define y4d(i3, i2, i1, i0) y[(i3) * (M * H_out * W_out) + (i2) * (H_out * W_out) + (i1) * (W_out) + i0]
    #define x4d(i3, i2, i1, i0) x[(i3) * (C * H * W) + (i2) * (H * W) + (i1) * (W) + i0]
    #define k4d(i3, i2, i1, i0) k[(i3) * (C * K * K) + (i2) * (K * K) + (i1) * (K) + i0]

    // Insert your GPU convolution kernel code here
    int w_grid = ceil(1.*W_out/TILE_WIDTH);
    int h_grid = ceil(1.*H_out/TILE_WIDTH);

    int n = blockIdx.x;
    int m = blockIdx.y;
    int h = blockIdx.z/w_grid*TILE_WIDTH + threadIdx.y;
    int w = blockIdx.z%w_grid*TILE_WIDTH + threadIdx.x;


    if ((w < (W_out)) && (h < (H_out))) {
        float acc = 0.0f;
        for (int c = 0; c<C; c++){          // loop all input channels
            for (int p = 0; p<K; p++){      // loop over k*k filter
                for (int q = 0; q<K; q++){
                    acc += x4d(n,c,h+p,w+q)*k4d(m,c,p,q);
                }
            }
        }
        y4d(n,m,h,w) = acc;
    }
    #undef y4d
    #undef x4d
    #undef k4d
}


// modified host code, using overlap to enlarge efficiency
__host__ void GPUInterface::conv_forward_gpu_prolog(float *host_y, const float *host_x, const float *host_k, float **device_y_ptr, float **device_x_ptr, float **device_k_ptr, const int B, const int M, const int C, const int H, const int W, const int K)
{
    // Allocate memory and copy over the relevant data structures to the GPU
    const int H_out = H - K + 1;
    const int W_out = W - K + 1;

    int stream_num = 5;
    int times = 10;
    int small_batch = B/stream_num/times;

    hipMalloc((void**) device_x_ptr, sizeof(float)*B*C*H*W);
    hipMalloc((void**) device_y_ptr, sizeof(float)*B*M*H_out*W_out);
    hipMalloc((void**) device_k_ptr, sizeof(float)*K*K*C*M);

    int w_grid = ceil(1.*W_out/TILE_WIDTH);
    int h_grid = ceil(1.*H_out/TILE_WIDTH);
    int Z = w_grid*h_grid;

    dim3 blockDim(TILE_WIDTH,TILE_WIDTH,1);
    dim3 gridDim(small_batch,M,Z);


    hipStream_t stream0,stream1,stream2,stream3,stream4,stream5,stream6,stream7,stream8,stream9;

    hipStreamCreate(&stream0);
    hipStreamCreate(&stream1);
    hipStreamCreate(&stream2);
    hipStreamCreate(&stream3);
    hipStreamCreate(&stream4);
    // hipStreamCreate(&stream5);
    // hipStreamCreate(&stream6);
    // hipStreamCreate(&stream7);
    // hipStreamCreate(&stream8);
    // hipStreamCreate(&stream9);

    for (int i=0; i<times; i++)
    {
        hipMemcpyAsync(*device_x_ptr + (0+stream_num*i)*small_batch*C*H*W, host_x+(0+stream_num*i)*small_batch*C*H*W, sizeof(float)*small_batch*C*H*W, hipMemcpyHostToDevice,stream0);
        hipMemcpyAsync(*device_x_ptr + (1+stream_num*i)*small_batch*C*H*W, host_x+(1+stream_num*i)*small_batch*C*H*W, sizeof(float)*small_batch*C*H*W, hipMemcpyHostToDevice,stream1);
        hipMemcpyAsync(*device_x_ptr + (2+stream_num*i)*small_batch*C*H*W, host_x+(2+stream_num*i)*small_batch*C*H*W, sizeof(float)*small_batch*C*H*W, hipMemcpyHostToDevice,stream2);
        hipMemcpyAsync(*device_x_ptr + (3+stream_num*i)*small_batch*C*H*W, host_x+(3+stream_num*i)*small_batch*C*H*W, sizeof(float)*small_batch*C*H*W, hipMemcpyHostToDevice,stream3);
        hipMemcpyAsync(*device_x_ptr + (4+stream_num*i)*small_batch*C*H*W, host_x+(4+stream_num*i)*small_batch*C*H*W, sizeof(float)*small_batch*C*H*W, hipMemcpyHostToDevice,stream4);
        // hipMemcpyAsync(*device_x_ptr + (5+stream_num*i)*small_batch*C*H*W, host_x+(5+stream_num*i)*small_batch*C*H*W, sizeof(float)*small_batch*C*H*W, hipMemcpyHostToDevice,stream5);
        // hipMemcpyAsync(*device_x_ptr + (6+stream_num*i)*small_batch*C*H*W, host_x+(6+stream_num*i)*small_batch*C*H*W, sizeof(float)*small_batch*C*H*W, hipMemcpyHostToDevice,stream6);
        // hipMemcpyAsync(*device_x_ptr + (7+stream_num*i)*small_batch*C*H*W, host_x+(7+stream_num*i)*small_batch*C*H*W, sizeof(float)*small_batch*C*H*W, hipMemcpyHostToDevice,stream7);
        // hipMemcpyAsync(*device_x_ptr + (8+stream_num*i)*small_batch*C*H*W, host_x+(8+stream_num*i)*small_batch*C*H*W, sizeof(float)*small_batch*C*H*W, hipMemcpyHostToDevice,stream8);
        // hipMemcpyAsync(*device_x_ptr + (9+stream_num*i)*small_batch*C*H*W, host_x+(9+stream_num*i)*small_batch*C*H*W, sizeof(float)*small_batch*C*H*W, hipMemcpyHostToDevice,stream9);
    
        conv_forward_kernel<<<gridDim,blockDim,0,stream0>>>(*device_y_ptr + (0+stream_num*i)*small_batch*M*H_out*W_out,*device_x_ptr + (0+stream_num*i)*small_batch*C*H*W,*device_k_ptr,small_batch,M,C,H,W,K);
        conv_forward_kernel<<<gridDim,blockDim,0,stream1>>>(*device_y_ptr + (1+stream_num*i)*small_batch*M*H_out*W_out,*device_x_ptr + (1+stream_num*i)*small_batch*C*H*W,*device_k_ptr,small_batch,M,C,H,W,K);
        conv_forward_kernel<<<gridDim,blockDim,0,stream2>>>(*device_y_ptr + (2+stream_num*i)*small_batch*M*H_out*W_out,*device_x_ptr + (2+stream_num*i)*small_batch*C*H*W,*device_k_ptr,small_batch,M,C,H,W,K);
        conv_forward_kernel<<<gridDim,blockDim,0,stream3>>>(*device_y_ptr + (3+stream_num*i)*small_batch*M*H_out*W_out,*device_x_ptr + (3+stream_num*i)*small_batch*C*H*W,*device_k_ptr,small_batch,M,C,H,W,K);
        conv_forward_kernel<<<gridDim,blockDim,0,stream4>>>(*device_y_ptr + (4+stream_num*i)*small_batch*M*H_out*W_out,*device_x_ptr + (4+stream_num*i)*small_batch*C*H*W,*device_k_ptr,small_batch,M,C,H,W,K);
        // conv_forward_kernel_shared_constant_multikernel<<<gridDim_choice,blockDim_choice,shemem_size_choice,stream5>>>(tile_size, *device_y_ptr + (5+stream_num*i)*small_batch*M*H_out*W_out,*device_x_ptr + (5+stream_num*i)*small_batch*C*H*W,*device_k_ptr,small_batch,M,C,H,W,K);
        // conv_forward_kernel_shared_constant_multikernel<<<gridDim_choice,blockDim_choice,shemem_size_choice,stream6>>>(tile_size, *device_y_ptr + (6+stream_num*i)*small_batch*M*H_out*W_out,*device_x_ptr + (6+stream_num*i)*small_batch*C*H*W,*device_k_ptr,small_batch,M,C,H,W,K);
        // conv_forward_kernel_shared_constant_multikernel<<<gridDim_choice,blockDim_choice,shemem_size_choice,stream7>>>(tile_size, *device_y_ptr + (7+stream_num*i)*small_batch*M*H_out*W_out,*device_x_ptr + (7+stream_num*i)*small_batch*C*H*W,*device_k_ptr,small_batch,M,C,H,W,K);
        // conv_forward_kernel_shared_constant_multikernel<<<gridDim_choice,blockDim_choice,shemem_size_choice,stream8>>>(tile_size, *device_y_ptr + (8+stream_num*i)*small_batch*M*H_out*W_out,*device_x_ptr + (8+stream_num*i)*small_batch*C*H*W,*device_k_ptr,small_batch,M,C,H,W,K);
        // conv_forward_kernel_shared_constant_multikernel<<<gridDim_choice,blockDim_choice,shemem_size_choice,stream9>>>(tile_size, *device_y_ptr + (9+stream_num*i)*small_batch*M*H_out*W_out,*device_x_ptr + (9+stream_num*i)*small_batch*C*H*W,*device_k_ptr,small_batch,M,C,H,W,K);
    
        hipMemcpyAsync(host_y+(0+stream_num*i)*small_batch*M*H_out*W_out, *device_y_ptr + (0+stream_num*i)*small_batch*M*H_out*W_out, sizeof(float)*small_batch*M*H_out*W_out,hipMemcpyDeviceToHost,stream0);
        hipMemcpyAsync(host_y+(1+stream_num*i)*small_batch*M*H_out*W_out, *device_y_ptr + (1+stream_num*i)*small_batch*M*H_out*W_out, sizeof(float)*small_batch*M*H_out*W_out,hipMemcpyDeviceToHost,stream1);
        hipMemcpyAsync(host_y+(2+stream_num*i)*small_batch*M*H_out*W_out, *device_y_ptr + (2+stream_num*i)*small_batch*M*H_out*W_out, sizeof(float)*small_batch*M*H_out*W_out,hipMemcpyDeviceToHost,stream2);
        hipMemcpyAsync(host_y+(3+stream_num*i)*small_batch*M*H_out*W_out, *device_y_ptr + (3+stream_num*i)*small_batch*M*H_out*W_out, sizeof(float)*small_batch*M*H_out*W_out,hipMemcpyDeviceToHost,stream3);
        hipMemcpyAsync(host_y+(4+stream_num*i)*small_batch*M*H_out*W_out, *device_y_ptr + (4+stream_num*i)*small_batch*M*H_out*W_out, sizeof(float)*small_batch*M*H_out*W_out,hipMemcpyDeviceToHost,stream4);
        // hipMemcpyAsync(host_y+(5+stream_num*i)*small_batch*M*H_out*W_out, *device_y_ptr + (5+stream_num*i)*small_batch*M*H_out*W_out, sizeof(float)*small_batch*M*H_out*W_out,hipMemcpyDeviceToHost,stream5);
        // hipMemcpyAsync(host_y+(6+stream_num*i)*small_batch*M*H_out*W_out, *device_y_ptr + (6+stream_num*i)*small_batch*M*H_out*W_out, sizeof(float)*small_batch*M*H_out*W_out,hipMemcpyDeviceToHost,stream6);
        // hipMemcpyAsync(host_y+(7+stream_num*i)*small_batch*M*H_out*W_out, *device_y_ptr + (7+stream_num*i)*small_batch*M*H_out*W_out, sizeof(float)*small_batch*M*H_out*W_out,hipMemcpyDeviceToHost,stream7);
        // hipMemcpyAsync(host_y+(8+stream_num*i)*small_batch*M*H_out*W_out, *device_y_ptr + (8+stream_num*i)*small_batch*M*H_out*W_out, sizeof(float)*small_batch*M*H_out*W_out,hipMemcpyDeviceToHost,stream8);
        // hipMemcpyAsync(host_y+(9+stream_num*i)*small_batch*M*H_out*W_out, *device_y_ptr + (9+stream_num*i)*small_batch*M*H_out*W_out, sizeof(float)*small_batch*M*H_out*W_out,hipMemcpyDeviceToHost,stream9);    
    }
 
    //Useful snippet for error checking
    hipError_t error = hipGetLastError();
    if(error != hipSuccess)
    {
        std::cout<<"CUDA error: "<<hipGetErrorString(error)<<std::endl;
        exit(-1);
    }

}


__host__ void GPUInterface::conv_forward_gpu(float *device_y, const float *device_x, const float *device_k, const int B, const int M, const int C, const int H, const int W, const int K)
{
    // Set the kernel dimensions and call the kernel
}


__host__ void GPUInterface::conv_forward_gpu_epilog(float *host_y, float *device_y, float *device_x, float *device_k, const int B, const int M, const int C, const int H, const int W, const int K)
{
    // Copy the output back to host
    hipFree(device_y);
    hipFree(device_x);
    hipFree(device_k);
}





__host__ void GPUInterface::get_device_properties()
{
    int deviceCount;
    hipGetDeviceCount(&deviceCount);

    for(int dev = 0; dev < deviceCount; dev++)
    {
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, dev);

        std::cout<<"Device "<<dev<<" name: "<<deviceProp.name<<std::endl;
        std::cout<<"Computational capabilities: "<<deviceProp.major<<"."<<deviceProp.minor<<std::endl;
        std::cout<<"Max Global memory size: "<<deviceProp.totalGlobalMem<<std::endl;
        std::cout<<"Max Constant memory size: "<<deviceProp.totalConstMem<<std::endl;
        std::cout<<"Max Shared memory size per block: "<<deviceProp.sharedMemPerBlock<<std::endl;
        std::cout<<"Max threads per block: "<<deviceProp.maxThreadsPerBlock<<std::endl;
        std::cout<<"Max block dimensions: "<<deviceProp.maxThreadsDim[0]<<" x, "<<deviceProp.maxThreadsDim[1]<<" y, "<<deviceProp.maxThreadsDim[2]<<" z"<<std::endl;
        std::cout<<"Max grid dimensions: "<<deviceProp.maxGridSize[0]<<" x, "<<deviceProp.maxGridSize[1]<<" y, "<<deviceProp.maxGridSize[2]<<" z"<<std::endl;
        std::cout<<"Warp Size: "<<deviceProp.warpSize<<std::endl;
    }
}
